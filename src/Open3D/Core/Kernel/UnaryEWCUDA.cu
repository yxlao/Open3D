// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include "Open3D/Core/Kernel/UnaryEW.h"

#include "Open3D/Core/CUDAState.cuh"
#include "Open3D/Core/CUDAUtils.h"
#include "Open3D/Core/Dispatch.h"
#include "Open3D/Core/Kernel/CUDALauncher.cuh"
#include "Open3D/Core/Tensor.h"

namespace open3d {
namespace kernel {

template <typename scalar_t>
static OPEN3D_HOST_DEVICE void CUDACopyElementKernel(const void* src,
                                                     void* dst) {
    *static_cast<scalar_t*>(dst) = *static_cast<const scalar_t*>(src);
}

void CopyCUDA(const Tensor& src, Tensor& dst) {
    // It has been checked that
    // - src and dst have the same shape, dtype
    // - at least one of src or dst is CUDA device
    SizeVector shape = src.GetShape();
    Dtype dtype = src.GetDtype();

    Device src_device = src.GetDevice();
    Device dst_device = dst.GetDevice();

    if (src_device.GetType() == Device::DeviceType::CUDA &&
        dst_device.GetType() == Device::DeviceType::CUDA) {
        if (src.IsContiguous() && dst.IsContiguous() &&
            src.GetShape() == dst.GetShape()) {
            // MemoryManager handles p2p and non-p2p device copy.
            MemoryManager::Memcpy(
                    dst.GetDataPtr(), dst_device, src.GetDataPtr(), src_device,
                    DtypeUtil::ByteSize(dtype) * shape.NumElements());
        } else if (src_device == dst_device) {
            // For more optimized version, one can check if P2P from src to dst
            // is enabled, then put synchronization with streams on both src and
            // dst to wait for copy kernel to complete.
            CUDADeviceSwitcher switcher(src_device);
            DISPATCH_DTYPE_TO_TEMPLATE(dtype, [&]() {
                Indexer indexer({src}, dst);
                CUDALauncher::LaunchUnaryEWKernel<scalar_t>(
                        indexer,
                        // Need to wrap as extended CUDA lamba function
                        [] OPEN3D_HOST_DEVICE(const void* src, void* dst) {
                            CUDACopyElementKernel<scalar_t>(src, dst);
                        });
            });
        } else {
            dst.CopyFrom(src.Contiguous().Copy(dst_device));
        }
    } else if (src_device.GetType() == Device::DeviceType::CPU &&
                       dst_device.GetType() == Device::DeviceType::CUDA ||
               src_device.GetType() == Device::DeviceType::CUDA &&
                       dst_device.GetType() == Device::DeviceType::CPU) {
        Tensor src_conti = src.Contiguous();  // No op if already contiguous
        if (dst.IsContiguous() && src.GetShape() == dst.GetShape()) {
            MemoryManager::Memcpy(
                    dst.GetDataPtr(), dst_device, src_conti.GetDataPtr(),
                    src_conti.GetDevice(),
                    DtypeUtil::ByteSize(dtype) * shape.NumElements());
        } else {
            dst.CopyFrom(src.Contiguous().Copy(dst_device));
        }
    } else {
        utility::LogError("Wrong device type {} -> {}", src_device.ToString(),
                          dst_device.ToString());
    }
}

}  // namespace kernel
}  // namespace open3d
